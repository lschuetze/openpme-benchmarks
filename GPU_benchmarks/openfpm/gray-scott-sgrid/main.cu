#include "hip/hip_runtime.h"
#include "Grid/grid_dist_id.hpp"
#include "data_type/aggregate.hpp"
#include "timer.hpp"

/*!
 *
 * \page Grid_3_gs_3D_sparse_gpu Gray Scott in 3D using sparse grids on GPU
 *
 * [TOC]
 *
 * # Solving a gray scott-system in 3D using Sparse grids on gpu # {#e3_gs_gray_scott_gpu}
 *
 * This example show how to solve a Gray-Scott system in 3D using sparse grids on gpu
 *
 * In figure is the final solution of the problem
 *
 * \htmlonly
 * <img src="http://ppmcore.mpi-cbg.de/web/images/examples/gray_scott_3d/gs_alpha.png"/>
 * \endhtmlonly
 *
 * More or less this example is the adaptation of the dense example in 3D
 *
 * \see \ref Grid_3_gs_3D
 *
 * # Initializetion
 *
 * On gpu we can add points using the function addPoints this function take 2 lamda functions the first take 3 arguments (in 3D)
 * i,j,k these are the global coordinates for a point. We can return either true either false. In case of true the point is
 * created in case of false the point is not inserted. The second lamda is instead used to initialize the point inserted.
 * The arguments of the second lambda are the data argument we use to initialize the point and the global coordinates i,j,k
 *
 * After we add the points we have to flush the added points. This us achieved using the function flush the template parameters
 * indicate how we have to act on the points. Consider infact we are adding points already exist ... do we have to add it using the max
 * or the min. **FLUSH_ON_DEVICE** say instead that the operation is performed using the GPU
 *
 * \snippet SparseGrid/1_gray_scott_3d_sparse_gpu/main.cu create points
 *
 * The function can also called with a specified range
 *
 * \snippet SparseGrid/1_gray_scott_3d_sparse_gpu/main.cu create points sub
 *
 * # Update
 *
 * to calculate the right-hand-side we use the function **conv2** this function can be used to do a convolution that involve
 * two properties
 *
 * The function accept a lambda function where the first 2 arguments are the output of the same type of the two property choosen.
 *
 * The arguments 3 and 4 contain the properties of two selected properties. while i,j,k are the coordinates we have to calculate the
 * convolution. The call **conv2** also accept template parameters the first two indicate the source porperties, the other two are the destination properties. While the
 * last is the extension of the stencil. In this case we use 1.
 *
 * The lambda function is defined as
 *
 * \snippet SparseGrid/1_gray_scott_3d_sparse_gpu/main.cu lambda
 *
 * and used in the body loop
 *
 * \snippet SparseGrid/1_gray_scott_3d_sparse_gpu/main.cu body
 *
 */

#ifdef __NVCC__

constexpr int U = 0;
constexpr int V = 1;

constexpr int U_next = 2;
constexpr int V_next = 3;

constexpr int x = 0;
constexpr int y = 1;
constexpr int z = 2;

typedef sgrid_dist_id_gpu<3,float,aggregate<float,float,float,float> > SparseGridType;

void init(SparseGridType & grid, Box<3,float> & domain)
{
	//! \cond [create points] \endcond

	typedef typename GetAddBlockType<SparseGridType>::type InsertBlockT;

	grid.addPoints([] __device__ (int i, int j, int k)
			        {
						return true;
			        },
			        [] __device__ (InsertBlockT & data, int i, int j, int k)
			        {
			        	data.template get<U>() = 1.0;
			        	data.template get<V>() = 0.0;
			        }
			        );


	grid.template flush<smax_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);

	//! \cond [create points] \endcond

	long int x_start = grid.size(0)*1.55f/domain.getHigh(0);
	long int y_start = grid.size(1)*1.55f/domain.getHigh(1);
	long int z_start = grid.size(1)*1.55f/domain.getHigh(2);

	long int x_stop = grid.size(0)*1.85f/domain.getHigh(0);
	long int y_stop = grid.size(1)*1.85f/domain.getHigh(1);
	long int z_stop = grid.size(1)*1.85f/domain.getHigh(2);

	//! \cond [create points sub] \endcond

	grid_key_dx<3> start({x_start,y_start,z_start});
	grid_key_dx<3> stop ({x_stop,y_stop,z_stop});

        grid.addPoints(start,stop,[] __device__ (int i, int j, int k)
                                {
                                                return true;
                                },
                                [] __device__ (InsertBlockT & data, int i, int j, int k)
                                {
                                        data.template get<U>() = 0.5;
                                        data.template get<V>() = 0.24;
                                }
                                );

	grid.template flush<smax_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);

	//! \cond [create points sub] \endcond
}


int main(int argc, char* argv[])
{
	openfpm_init(&argc,&argv);

	// domain
	Box<3,float> domain({0.0,0.0,0.0},{2.5,2.5,2.5});
	
	// grid size
        size_t sz[3] = {256,256,256};

	// Define periodicity of the grid
	periodicity<3> bc = {PERIODIC,PERIODIC,PERIODIC};
	
	// Ghost in grid unit
	Ghost<3,long int> g(1);
	
	// deltaT
	float deltaT = 0.25;

	// Diffusion constant for specie U
	float du = 2*1e-5;

	// Diffusion constant for specie V
	float dv = 1*1e-5;

	// Number of timesteps
#ifdef TEST_RUN
	size_t timeSteps = 300;
#else
        size_t timeSteps = 15000;
#endif

	// K and F (Physical constant in the equation)
    float K = 0.053;
    float F = 0.014;

	sgrid_dist_id_gpu<3, float, aggregate<float,float,float,float>> grid(sz,domain,g,bc);

	// spacing of the grid on x and y
	float spacing[3] = {grid.spacing(0),grid.spacing(1),grid.spacing(2)};

	init(grid,domain);

	// sync the ghost
	grid.template ghost_get<U,V>(RUN_ON_DEVICE);

	// because we assume that spacing[x] == spacing[y] we use formula 2
	// and we calculate the prefactor of Eq 2
	float uFactor = deltaT * du/(spacing[x]*spacing[x]);
	float vFactor = deltaT * dv/(spacing[x]*spacing[x]);

	auto & v_cl = create_vcluster();

	timer tot_sim;
	tot_sim.start();

	for (size_t i = 0; i < timeSteps ; ++i)
	{
		if (v_cl.rank() == 0)
		{std::cout << "STEP: " << i << std::endl;}
/*		if (i % 300 == 0)
		{
			std::cout << "STEP: " << i << std::endl;
			grid.write_frame("out",i,VTK_WRITER);
		}*/

		//! \cond [stencil get and use] \endcond

		typedef typename GetCpBlockType<decltype(grid),0,1>::type CpBlockType;

		//! \cond [lambda] \endcond

		auto func = [uFactor,vFactor,deltaT,F,K] __device__ (float & u_out, float & v_out,
				                                   CpBlockType & u, CpBlockType & v,
				                                   int i, int j, int k){

				float uc = u(i,j,k);
				float vc = v(i,j,k);

				u_out = uc + uFactor *(u(i-1,j,k) + u(i+1,j,k) +
                                                       u(i,j-1,k) + u(i,j+1,k) +
                                                       u(i,j,k-1) + u(i,j,k+1) - 6.0*uc) - deltaT * uc*vc*vc
                                                       - deltaT * F * (uc - 1.0);


				v_out = vc + vFactor *(v(i-1,j,k) + v(i+1,j,k) +
                                                       v(i,j+1,k) + v(i,j-1,k) +
                                                       v(i,j,k-1) + v(i,j,k+1) - 6.0*vc) + deltaT * uc*vc*vc
					               - deltaT * (F+K) * vc;
				};

		//! \cond [lambda] \endcond

		//! \cond [body] \endcond

		if (i % 2 == 0)
		{
			grid.conv2<U,V,U_next,V_next,1>({0,0,0},{(long int)sz[0]-1,(long int)sz[1]-1,(long int)sz[2]-1},func);

			// After copy we synchronize again the ghost part U and V

			grid.ghost_get<U_next,V_next>(RUN_ON_DEVICE | SKIP_LABELLING);
		}
		else
		{
			grid.conv2<U_next,V_next,U,V,1>({0,0,0},{(long int)sz[0]-1,(long int)sz[1]-1,(long int)sz[2]-1},func);

			// After copy we synchronize again the ghost part U and V
			grid.ghost_get<U,V>(RUN_ON_DEVICE | SKIP_LABELLING);
		}

		//! \cond [body] \endcond

		// Every 500 time step we output the configuration for
		// visualization
//		if (i % 500 == 0)
//		{
//			grid.save("output_" + std::to_string(count));
//			count++;
//		}
	}
	
	tot_sim.stop();
	std::cout << "Total simulation: " << tot_sim.getwct() << std::endl;

	grid.deviceToHost<U,V,U_next,V_next>();
	grid.write("final");

	//! \cond [time stepping] \endcond

	/*!
	 * \page Grid_3_gs_3D_sparse Gray Scott in 3D
	 *
	 * ## Finalize ##
	 *
	 * Deinitialize the library
	 *
	 * \snippet Grid/3_gray_scott/main.cpp finalize
	 *
	 */

	//! \cond [finalize] \endcond

	openfpm_finalize();

	//! \cond [finalize] \endcond

	/*!
	 * \page Grid_3_gs_3D_sparse Gray Scott in 3D
	 *
	 * # Full code # {#code}
	 *
	 * \include Grid/3_gray_scott_3d/main.cpp
	 *
	 */
}

#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif

