#include "hip/hip_runtime.h"
//#define VCLUSTER_PERF_REPORT
//#define SYNC_BEFORE_TAKE_TIME
//#define ENABLE_GRID_DIST_ID_PERF_STATS
#include "Grid/grid_dist_id.hpp"
#include "data_type/aggregate.hpp"
#include "timer.hpp"

/*!
 *
 * \page Grid_3_gs_3D_sparse_gpu_cs_opt Gray Scott in 3D using sparse grids on gpu in complex geometry
 *
 * [TOC]
 *
 * # Solving a gray scott-system in 3D using Sparse grids# {#e3_gs_gray_scott}
 *
 * This example show how to solve a Gray-Scott system in 3D using sparse grids on gpu with complex geometry
 *
 * In figure is the final solution of the problem
 *
 * \htmlonly
<table border="1" bgcolor="black">
  <tr>
    <td>
      <img src="http://ppmcore.mpi-cbg.de/web/images/examples/1_gray_scott_3d_sparse_cs/gs_3d_sparse_cs_section.png" style="width: 500px;" />
    </td>
    <td>
      <img src="http://ppmcore.mpi-cbg.de/web/images/examples/1_gray_scott_3d_sparse_cs/gs_3d_sparse_cs.png" style="width: 500px;" />
    </td>
  </tr>
</table>
\endhtmlonly
 *
 * More or less this example is the same of \ref e3_gs_gray_scott_cs on gpu using what we learned in \ref e3_gs_gray_scott_gpu
 *
 *
 */

#ifdef __NVCC__

constexpr int U = 0;
constexpr int V = 1;
constexpr int U_next = 2;
constexpr int V_next = 3;

typedef sgrid_dist_id_gpu<3,double,aggregate<double,double,double,double> > sgrid_type;

void init(sgrid_type & grid, Box<3,double> & domain)
{
	auto it = grid.getGridIterator();
	Point<3,double> p[8]= {{0.35,0.35,0.35},
	                       {0.35,2.0,2.0},
	                       {2.0,0.35,2.0},
	                       {2.0,2.0,0.35},
	                       {0.35,0.35,2.0},
	                       {0.35,2.0,0.35},
			       {2.0,0.35,0.35},
	                       {2.0,2.0,2.0}};

	
//	Point<3,double> u({1.0,0.0,0.0});
//	Box<3,double> channel_box(p3,p1);

	double spacing_x = grid.spacing(0);
	double spacing_y = grid.spacing(1);
	double spacing_z = grid.spacing(2);

	typedef typename GetAddBlockType<sgrid_type>::type InsertBlockT;

	// Draw spheres
	for (int i = 0 ; i < 8 ; i++)
	{
		Sphere<3,double> sph(p[i],0.3);

		Box<3,size_t> bx;

		for (int i = 0 ; i < 3 ; i++)
		{
			bx.setLow(i,(size_t)((sph.center(i) - 0.31)/grid.spacing(i)));
			bx.setHigh(i,(size_t)((sph.center(i) + 0.31)/grid.spacing(i)));
		}

		grid.addPoints(bx.getKP1(),bx.getKP2(),[spacing_x,spacing_y,spacing_z,sph] __device__ (int i, int j, int k)
                                {
                                                Point<3,double> pc({i*spacing_x,j*spacing_y,k*spacing_z});

						// Check if the point is in the domain
                                		if (sph.isInside(pc) )
                                		{return true;}

                                                return false;
                                },
                                [] __device__ (InsertBlockT & data, int i, int j, int k)
                                {
                                        data.template get<U>() = 1.0;
                                        data.template get<V>() = 0.0;
                                }
                                );

		grid.template flush<smax_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);
		grid.removeUnusedBuffers();
	}

	//channels

	Box<3,double> b({0.25,0.25,0.25},{2.1,2.1,2.1});

	for (int k = 0 ; k < 3 ; k++)
	{
		for (int s = 0 ; s < 2 ; s++)
		{
			for (int i = 0 ; i < 2 ; i++)
        		{
				Point<3,double> u({1.0*(((s+i)%2) == 0 && k != 2),1.0*(((s+i+1)%2) == 0 && k != 2),(k == 2)*1.0});
				Point<3,double> c({(i == 0)?0.35:2.0,(s == 0)?0.35:2.0,(k == 0)?0.35:2.0});

                		Box<3,size_t> bx;

                		for (int i = 0 ; i < 3 ; i++)
                		{
					if (c[i] == 2.0)
					{
						if (u[i] == 1.0)
						{
                                                	bx.setLow(i,(size_t)(0.34/grid.spacing(i)));
                                                	bx.setHigh(i,(size_t)(2.01/grid.spacing(i)));
						}
						else
						{
                                                        bx.setLow(i,(size_t)((c[i] - 0.11)/grid.spacing(i)));
                                                        bx.setHigh(i,(size_t)((c[i] + 0.11)/grid.spacing(i)));
						}
					}
					else
					{
						if (u[i] == 1.0)
						{
                        				bx.setLow(i,(size_t)(0.34/grid.spacing(i)));
                        				bx.setHigh(i,(size_t)(2.01/grid.spacing(i)));
						}
						else
						{
                                                        bx.setLow(i,(size_t)((c[i] - 0.11)/grid.spacing(i)));
                                                        bx.setHigh(i,(size_t)((c[i] + 0.11)/grid.spacing(i)));
						}
					}
                		}

				grid.addPoints(bx.getKP1(),bx.getKP2(),[spacing_x,spacing_y,spacing_z,u,c,b] __device__ (int i, int j, int k)
                                	{
						Point<3,double> pc({i*spacing_x,j*spacing_y,k*spacing_z});
                                                Point<3,double> pcs({i*spacing_x,j*spacing_y,k*spacing_z});
                                                Point<3,double> vp;

						// shift
						pc -= c; 

                                		// calculate the distance from the diagonal
                                		vp.get(0) = pc.get(1)*u.get(2) - pc.get(2)*u.get(1);
                                		vp.get(1) = pc.get(2)*u.get(0) - pc.get(0)*u.get(2);
                                		vp.get(2) = pc.get(0)*u.get(1) - pc.get(1)*u.get(0);

						double distance = vp.norm();

                                                // Check if the point is in the domain
                                                if (distance < 0.1 && b.isInside(pcs) == true )
                                                {return true;}

                                                return false;
                                	},
                                	[] __device__ (InsertBlockT & data, int i, int j, int k)
                                	{
                                        	data.template get<U>() = 1.0;
                                        	data.template get<V>() = 0.0;
                                	}
                                );

                		grid.template flush<smax_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);
				grid.removeUnusedBuffers();
			}
		}
	}

	// cross channel
	
	int s = 0;
	for (int s = 0 ; s < 2 ; s++)
        {
        	for (int i = 0 ; i < 2 ; i++)
                {	
			Point<3,double> c({(i == 0)?0.35:2.0,(s == 0)?0.35:2.0,0.35});
			Point<3,double> u({(i == 0)?1.0:-1.0,(s == 0)?1.0:-1.0,1.0});

			Box<3,size_t> bx;

			for (int k = 0 ; k < 16; k++)
			{
				for (int s = 0 ; s < 3 ; s++)
				{
					if (u[s] > 0.0)
					{
						bx.setLow(s,(c[s] + k*(u[s]/9.0))/grid.spacing(s) );
						bx.setHigh(s,(c[s] + (k+3)*(u[s]/9.0) )/ grid.spacing(s) );
					}
					else
					{
						bx.setLow(s,(c[s] + (k+3)*(u[s]/9.0) )/grid.spacing(s) );
                                                bx.setHigh(s,(c[s] + k*(u[s]/9.0))/ grid.spacing(s) );
					}
				}

				grid.addPoints(bx.getKP1(),bx.getKP2(),[spacing_x,spacing_y,spacing_z,u,c,b] __device__ (int i, int j, int k)
        			{
                			Point<3,double> pc({i*spacing_x,j*spacing_y,k*spacing_z});
                        		Point<3,double> pcs({i*spacing_x,j*spacing_y,k*spacing_z});
                        		Point<3,double> vp;

                        		// shift
                        		pc -= c;

                        		// calculate the distance from the diagonal
                        		vp.get(0) = pc.get(1)*u.get(2) - pc.get(2)*u.get(1);
                        		vp.get(1) = pc.get(2)*u.get(0) - pc.get(0)*u.get(2);
                        		vp.get(2) = pc.get(0)*u.get(1) - pc.get(1)*u.get(0);

                        		double distance = vp.norm() / sqrt(3.0);

                        		// Check if the point is in the domain
                        		if (distance < 0.1 && b.isInside(pcs) == true )
                        		{return true;}

                        		return false;
                  		},
                  		[] __device__ (InsertBlockT & data, int i, int j, int k)
                  		{
                  			data.template get<U>() = 1.0;
                        		data.template get<V>() = 0.0;
                  		}
        			);

				grid.template flush<smax_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);
				grid.removeUnusedBuffers();
			}
		}

	}

	long int x_start = grid.size(0)*1.95f/domain.getHigh(0);
	long int y_start = grid.size(1)*1.95f/domain.getHigh(1);
	long int z_start = grid.size(1)*1.95f/domain.getHigh(2);

	long int x_stop = grid.size(0)*2.05f/domain.getHigh(0);
	long int y_stop = grid.size(1)*2.05f/domain.getHigh(1);
	long int z_stop = grid.size(1)*2.05f/domain.getHigh(2);

	grid_key_dx<3> start({x_start,y_start,z_start});
	grid_key_dx<3> stop ({x_stop,y_stop,z_stop});

        grid.addPoints(start,stop,[] __device__ (int i, int j, int k)
                                {
                                                return true;
                                },
                                [] __device__ (InsertBlockT & data, int i, int j, int k)
                                {
                                        data.template get<U>() = 0.5;
                                        data.template get<V>() = 0.24;
                                }
                                );

	grid.template flush<smin_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);

	grid.removeUnusedBuffers();
}


int main(int argc, char* argv[])
{
	openfpm_init(&argc,&argv);

	// domain
	Box<3,double> domain({0.0,0.0,0.0},{2.5,2.5,2.5});
	
	// grid size
        size_t sz[3] = {384,384,384};

	// Define periodicity of the grid
	periodicity<3> bc = {NON_PERIODIC,NON_PERIODIC,NON_PERIODIC};
	
	// Ghost in grid unit
	Ghost<3,long int> g(1);
	
	// deltaT
	double deltaT = 0.2;

	// Diffusion constant for specie U
	double du = 2*1e-5;

	// Diffusion constant for specie V
	double dv = 1*1e-5;

#ifdef TEST_RUN
        // Number of timesteps
        size_t timeSteps = 300;
#else
	// Number of timesteps
        size_t timeSteps = 50000;
#endif

	// K and F (Physical constant in the equation)
        double K = 0.053;
        double F = 0.014;

	sgrid_type grid(sz,domain,g,bc);

	grid.template setBackgroundValue<0>(-0.5);
	grid.template setBackgroundValue<1>(-0.5);
	grid.template setBackgroundValue<2>(-0.5);
	grid.template setBackgroundValue<3>(-0.5);
	
	// spacing of the grid on x and y
	double spacing[3] = {grid.spacing(0),grid.spacing(1),grid.spacing(2)};

	init(grid,domain);

	// sync the ghost
	grid.template ghost_get<U,V>(RUN_ON_DEVICE);

	// because we assume that spacing[x] == spacing[y] we use formula 2
	// and we calculate the prefactor of Eq 2
	double uFactor = deltaT * du/(spacing[0]*spacing[0]);
	double vFactor = deltaT * dv/(spacing[0]*spacing[0]);

	grid.template deviceToHost<U,V>();

	timer tot_sim;
	tot_sim.start();

	for (size_t i = 0; i < timeSteps; ++i)
	{
		//! \cond [stencil get and use] \endcond

        		typedef typename GetCpBlockType<decltype(grid),0,1>::type CpBlockType;

        		auto func = [uFactor,vFactor,deltaT,F,K] __device__ (double & u_out, double & v_out,
        				                                   CpBlockType & u, CpBlockType & v,
        				                                   int i, int j, int k){

        				double uc = u(i,j,k);
        				double vc = v(i,j,k);

        				double u_px = u(i+1,j,k);
        				double u_mx = u(i-1,j,k);

        				double u_py = u(i,j+1,k);
        				double u_my = u(i,j-1,k);

        				double u_pz = u(i,j,k+1);
        				double u_mz = u(i,j,k-1);

        				double v_px = v(i+1,j,k);
        				double v_mx = v(i-1,j,k);

        				double v_py = v(i,j+1,k);
        				double v_my = v(i,j-1,k);

        				double v_pz = v(i,j,k+1);
        				double v_mz = v(i,j,k-1);

        				// U fix

        				if (u_mx < -0.1 && u_px < -0.1)
        				{
        					u_mx = uc;
        					u_px = uc;
        				}

        				if (u_mx < -0.1)
        				{u_mx = u_px;}

        				if (u_px < -0.1)
        				{u_px = u_mx;}

        				if (u_my < -0.1 && u_py < -0.1)
        				{
        					u_my = uc;
        					u_py = uc;
        				}

        				if (u_my < -0.1)
        				{u_my = u_py;}

        				if (u_py < -0.1)
        				{u_py = u_my;}

        				if (u_mz < -0.1 && u_pz < -0.1)
        				{
        					u_mz = uc;
        					u_pz = uc;
        				}

        				if (u_mz < -0.1)
        				{u_mz = u_pz;}

        				if (u_pz < -0.1)
        				{u_pz = u_mz;}

        				// V fix

        				if (v_mx < -0.1 && v_px < -0.1)
        				{
        					v_mx = uc;
        					v_px = uc;
        				}

        				if (v_mx < -0.1)
        				{v_mx = v_px;}

        				if (v_px < -0.1)
        				{v_px = v_mx;}

        				if (v_my < -0.1 && v_py < -0.1)
        				{
        					v_my = uc;
        					v_py = uc;
        				}

        				if (v_my < -0.1)
        				{v_my = v_py;}

        				if (v_py < -0.1)
        				{v_py = v_my;}

        				if (v_mz < -0.1 && v_pz < -0.1)
        				{
        					v_mz = uc;
        					v_pz = uc;
        				}

        				if (v_mz < -0.1)
        				{v_mz = v_pz;}

        				if (v_pz < -0.1)
        				{v_pz = v_mz;}

        				u_out = uc + uFactor *(u_mx + u_px +
                                                               u_my + u_py +
                                                               u_mz + u_pz - 6.0*uc) - deltaT * uc*vc*vc
                                                               - deltaT * F * (uc - 1.0);


        				v_out = vc + vFactor *(v_mx + v_px +
                                                               v_py + v_my +
                                                               v_mz + v_pz - 6.0*vc) + deltaT * uc*vc*vc
        					               - deltaT * (F+K) * vc;

        				};

        		if (i % 2 == 0)
        		{
        			grid.conv2<U,V,U_next,V_next,1>({0,0,0},{(long int)sz[0]-1,(long int)sz[1]-1,(long int)sz[2]-1},func);

				hipDeviceSynchronize();

        			// After copy we synchronize again the ghost part U and V

        			grid.ghost_get<U_next,V_next>(RUN_ON_DEVICE | SKIP_LABELLING);
        		}
        		else
        		{
        			grid.conv2<U_next,V_next,U,V,1>({0,0,0},{(long int)sz[0]-1,(long int)sz[1]-1,(long int)sz[2]-1},func);

				hipDeviceSynchronize();

        			// After copy we synchronize again the ghost part U and V
        			grid.ghost_get<U,V>(RUN_ON_DEVICE | SKIP_LABELLING);
        		}

		//! \cond [stencil get and use] \endcond

		// After copy we synchronize again the ghost part U and V

		// Every 500 time step we output the configuration for
		// visualization
/*		if (i % 500 == 0)
		{
			grid.save("output_" + std::to_string(count));
			count++;
		}*/

                std::cout << "STEP: " << i  << std::endl;
/*                if (i % 300 == 0)
                {
                	grid.template deviceToHost<U,V>();
                        grid.write_frame("out",i);
                }*/
	}
	
	tot_sim.stop();
	std::cout << "Total simulation: " << tot_sim.getwct() << std::endl;

	grid.print_stats();

	create_vcluster().print_stats();

	grid.template deviceToHost<U,V>();
	grid.write("Final");

	//! \cond [time stepping] \endcond

	/*!
	 * \page Grid_3_gs_3D_sparse_gpu_cs Gray Scott in 3D using sparse grids on gpu in complex geometry
	 *
	 * ## Finalize ##
	 *
	 * Deinitialize the library
	 *
	 * \snippet  SparseGrid/1_gray_scott_3d_sparse_gpu_cs/main.cu finalize
	 *
	 */

	//! \cond [finalize] \endcond

	openfpm_finalize();

	//! \cond [finalize] \endcond

	/*!
	 * \page Grid_3_gs_3D_sparse_gpu_cs Gray Scott in 3D using sparse grids on gpu in complex geometry
	 *
	 * # Full code # {#code}
	 *
	 * \include SparseGrid/1_gray_scott_3d_sparse_gpu_cs/main.cu
	 *
	 */
}

#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif

